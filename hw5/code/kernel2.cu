#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line)
{
    if (result)
    {
        printf("CUDA error %s at %s:%d '%s' \n", hipGetErrorString(result), file, line, func);

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        {
            break;
        }

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, size_t width, int *result, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    // size_t index = thisY * width + thisX;
    int result_ = mandel(x, y, maxIterations);

    // result[index] = result_;
    ((int *)((char *)result + thisY * width))[thisX] = result_;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate result array on host memory
    int *result = nullptr;
    // checkCudaErrors(cudaHostAlloc(&result, resX * resY * sizeof(int), cudaHostAllocMapped));
    checkCudaErrors(hipHostAlloc(&result, resX * resY * sizeof(int), hipHostMallocDefault));

    // Get the pointer to mapped memory on device
    int *cudaResult = nullptr;
    size_t pitch;
    // checkCudaErrors(cudaHostGetDevicePointer(&cudaResult, result, 0));
    checkCudaErrors(hipMallocPitch(&cudaResult, &pitch, resX * sizeof(int), resY));

    // 1600 x 1200 = 1920000
    dim3 blockSize(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 gridSize(resX / blockSize.x, resY / blockSize.y);

    mandelKernel<<<gridSize, blockSize>>>(lowerX, lowerY, stepX, stepY, pitch, cudaResult, maxIterations);

    // Copy result array from device to host memory
    checkCudaErrors(hipMemcpy(result, cudaResult, resX * resY * sizeof(int), hipMemcpyDeviceToHost));
    // checkCudaErrors(cudaMemcpy2D(result, resX * sizeof(int), cudaResult, pitch, resX * sizeof(int), resY, cudaMemcpyDeviceToHost));
    checkCudaErrors(hipFree(cudaResult));

    // // Change to use cudaMemcpy with cudaMemcpyHostToHost flag without calling another synchronization
    // checkCudaErrors(cudaMemcpy(img, result, resX * resY * sizeof(int), cudaMemcpyHostToHost));
    memcpy(img, result, resX * resY * sizeof(int));
    checkCudaErrors(hipHostFree(result));
}
