#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *result, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate result array on host memory
    int *result = (int *)malloc(resX * resY * sizeof(int));

    // Allocate result array on device memory
    int *cudaResult = NULL;
    hipMalloc(&cudaResult, resX * resY * sizeof(int));

    int gridSize;
    int blockSize;

    mandelKernel<<<1, 1>>>(lowerX, lowerY, stepX, stepY, cudaResult, maxIterations);

    // Copy result array from device to host memory
    hipMemcpy(result, cudaResult, resX * resY * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
    hipFree(cudaResult);

    memcpy(img, result, resX * resY * sizeof(int));
    free(result);
}
