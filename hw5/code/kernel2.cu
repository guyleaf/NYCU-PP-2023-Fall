#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

// comment out if you need faster implementation or define macro from outside
// #define USE_FASTER

// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line)
{
    if (result)
    {
        printf("CUDA error %s at %s:%d '%s' \n", hipGetErrorString(result), file, line, func);

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        {
            break;
        }

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, size_t width, int *result, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int result_ = mandel(x, y, maxIterations);

#ifdef USE_FASTER
    size_t index = thisY * width + thisX;
    result[index] = result_;
#else
    ((int *)((char *)result + thisY * width))[thisX] = result_;
#endif
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int *result = nullptr, *cudaResult = nullptr;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Plan 1: cudaHostAlloc + mapped (faster)
    // Plan 2: cudaHostAlloc + cudaMallocPitch + cudaMemcpy2D (slower) (homework required)

#ifdef USE_FASTER
    // Allocate result array on host memory
    checkCudaErrors(cudaHostAlloc(&result, resX * resY * sizeof(int), cudaHostAllocMapped));

    // Get the pointer to mapped memory on device
    checkCudaErrors(cudaHostGetDevicePointer(&cudaResult, result, 0));
#else
    // Allocate result array on host memory
    checkCudaErrors(hipHostAlloc(&result, resX * resY * sizeof(int), hipHostMallocDefault));

    size_t pitch;
    checkCudaErrors(hipMallocPitch(&cudaResult, &pitch, resX * sizeof(int), resY));
#endif

    // 1600 x 1200 = 1920000
    int block_width = BLOCK_WIDTH;
    int block_height = BLOCK_HEIGHT;
    dim3 blockSize(block_width, block_height);
    dim3 gridSize(resX / block_width, resY / block_height);

#ifdef USE_FASTER
    mandelKernel<<<gridSize, blockSize>>>(lowerX, lowerY, stepX, stepY, resX, cudaResult, maxIterations);

    // Use cudaMemcpy with cudaMemcpyHostToHost flag without calling another synchronization
    checkCudaErrors(cudaMemcpy(img, result, resX * resY * sizeof(int), cudaMemcpyHostToHost));
#else
    mandelKernel<<<gridSize, blockSize>>>(lowerX, lowerY, stepX, stepY, pitch, cudaResult, maxIterations);

    // Copy result array from device to host memory
    checkCudaErrors(hipMemcpy2D(result, resX * sizeof(int), cudaResult, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(cudaResult));

    memcpy(img, result, resX * resY * sizeof(int));
#endif

    checkCudaErrors(hipHostFree(result));
}
