#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

#define GROUP_WIDTH 8
#define GROUP_HEIGHT 8

// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line)
{
    if (result)
    {
        printf("CUDA error %s at %s:%d '%s' \n", hipGetErrorString(result), file, line, func);

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        {
            break;
        }

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int height, size_t pitch, int *result, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_WIDTH;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * GROUP_HEIGHT;

    width = min(thisX + GROUP_WIDTH, width);
    height = min(thisY + GROUP_HEIGHT, height);

    float x, y;
    for (size_t localY = thisY; localY < height; localY++)
    {
        y = lowerY + localY * stepY;
        for (size_t localX = thisX; localX < width; localX++)
        {
            x = lowerX + localX * stepX;
            ((int *)((char *)result + localY * pitch))[localX] = mandel(x, y, maxIterations);
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *result = nullptr;
    int *cudaResult = nullptr;
    size_t pitch;

    // Allocate result array on host memory
    checkCudaErrors(hipHostAlloc(&result, resX * resY * sizeof(int), hipHostMallocDefault));

    // Allocate padded array on device memory
    checkCudaErrors(hipMallocPitch(&cudaResult, &pitch, resX * sizeof(int), resY));

    // 1600 x 1200 = 1920000
    int block_width = BLOCK_WIDTH;
    int block_height = BLOCK_HEIGHT;
    dim3 blockSize(block_width, block_height);

    block_width *= GROUP_WIDTH;
    block_height *= GROUP_HEIGHT;
    dim3 gridSize((int)std::ceil((float)resX / block_width), (int)std::ceil((float)resY / block_height));

    mandelKernel<<<gridSize, blockSize>>>(lowerX, lowerY, stepX, stepY, resX, resY, pitch, cudaResult, maxIterations);

    // Copy result array from device to host memory
    checkCudaErrors(hipMemcpy2D(result, resX * sizeof(int), cudaResult, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(cudaResult));

    memcpy(img, result, resX * resY * sizeof(int));
    checkCudaErrors(hipHostFree(result));
}
