#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line)
{
    if (result)
    {
        printf("CUDA error %s at %s:%d '%s' \n", hipGetErrorString(result), file, line, func);

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f) break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int *result, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int index = (thisY * width + thisX);
    result[index] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Allocate result array on host memory
    int *result = (int *)malloc(resX * resY * sizeof(int));

    // Allocate result array on device memory
    int *cudaResult = NULL;
    checkCudaErrors(hipMalloc(&cudaResult, resX * resY * sizeof(int)));

    // 1600 x 1200 = 1920000
    dim3 blockSize(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 gridSize(resX / blockSize.x, resY / blockSize.y);

    mandelKernel<<<gridSize, blockSize>>>(lowerX, lowerY, stepX, stepY, resX, cudaResult, maxIterations);

    // Copy result array from device to host memory
    checkCudaErrors(hipMemcpy(result, cudaResult, resX * resY * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(cudaResult));

    memcpy(img, result, resX * resY * sizeof(int));
    free(result);
}
