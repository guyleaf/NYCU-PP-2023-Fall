#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

// comment out if you need faster implementation or define macro from outside
// #define USE_FASTER

// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line)
{
    if (result)
    {
        printf("CUDA error %s at %s:%d '%s' \n", hipGetErrorString(result), file, line, func);

        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        {
            break;
        }

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, size_t width, int *result, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int result_ = mandel(x, y, maxIterations);
    ((int *)((char *)result + thisY * width))[thisX] = result_;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    size_t pitch;
    int *cudaResult = nullptr;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Plan 1: cudaMallocPitch + cudaMemcpy2D to img directly (faster)
    // Plan 2: cudaHostAlloc + cudaMallocPitch + cudaMemcpy2D (slower) (homework required)

#ifdef USE_FASTER
    // Allocate padded array on device memory
    checkCudaErrors(cudaMallocPitch(&cudaResult, &pitch, resX * sizeof(int), resY));
#else
    int *result = nullptr;

    // Allocate result array on host memory
    checkCudaErrors(hipHostAlloc(&result, resX * resY * sizeof(int), hipHostMallocDefault));

    // Allocate padded array on device memory
    checkCudaErrors(hipMallocPitch(&cudaResult, &pitch, resX * sizeof(int), resY));
#endif

    // 1600 x 1200 = 1920000
    int block_width = BLOCK_WIDTH;
    int block_height = BLOCK_HEIGHT;
    dim3 blockSize(block_width, block_height);
    dim3 gridSize(resX / block_width, resY / block_height);

    mandelKernel<<<gridSize, blockSize>>>(lowerX, lowerY, stepX, stepY, pitch, cudaResult, maxIterations);

#ifdef USE_FASTER
    // Copy result array from device to host memory
    checkCudaErrors(cudaMemcpy2D(img, resX * sizeof(int), cudaResult, pitch, resX * sizeof(int), resY, cudaMemcpyDeviceToHost));
    checkCudaErrors(cudaFree(cudaResult));
#else
    // Copy result array from device to host memory
    checkCudaErrors(hipMemcpy2D(result, resX * sizeof(int), cudaResult, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(cudaResult));

    memcpy(img, result, resX * resY * sizeof(int));
    checkCudaErrors(hipHostFree(result));
#endif
}
